
#include <hip/hip_runtime.h>

__global__ void init(int ntx, int nty_local, int nWorkers, double *u) {


    int tid;
    int j, jj1, l;

    tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < nWorkers) {

        for(j = 0; j < nty_local; j++) {

            jj1 = j * ntx;

            for(l = 0; l < ntx; l++) {

                u[l + jj1] = 0.0;

            }
        }

        tid += blockDim.x * gridDim.x;
    }
}
